#include "hip/hip_runtime.h"
//
// Created by mohamadbastin on 7/7/22.
//
#include "cstdio"
#include "../common/common.cpp"

__global__ void matrixAdd2d1D(float *A, float *B, float *C, int nx, int ny) {

    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }

}

int main() {
    int nx = 64;
    int ny = 64;

    int nxy = nx * ny;
    int bytes = nxy * sizeof(float);

    float *h_A, *h_B, *h_C, *gpu_ref;
    h_A = (float *) malloc(bytes);
    h_B = (float *) malloc(bytes);
    h_C = (float *) malloc(bytes);
    gpu_ref = (float *) malloc(bytes);

    initialize(h_A, nxy);
    initialize(h_B, nxy);

    for (int i = 0; i < nxy; i++) {
        h_C[i] = h_A[i] + h_B[i];
//        printf("%f ", h_C[i]);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, bytes);
    hipMalloc((void **) &d_B, bytes);
    hipMalloc((void **) &d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    matrixAdd2d1D<<<grid, block>>>(d_A, d_B, d_C, nx, ny);
    hipDeviceSynchronize();

    hipMemcpy(gpu_ref, d_C, bytes, hipMemcpyDeviceToHost);

    int res = checkResult(h_C, gpu_ref, nxy);

    if (res == 1) {
        printf("success.\n");
        return 0;
    }
    printf("failed.\n");

    return 0;
}