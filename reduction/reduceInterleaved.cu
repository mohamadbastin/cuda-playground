#include "hip/hip_runtime.h"
//
// Created by mohamadbastin on 7/27/22.
//
#include "cstdio"
#include "../common/common.cpp"


__global__ void reduceInterleaved(int *g_inputData, int *g_outputData, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int bidx = blockDim.x * blockIdx.x;

    if (bidx + tid >= n) return;
    int *idata = bidx + g_inputData;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    if (tid == 0) {
        g_outputData[blockIdx.x] = idata[0];
    }
}

int main() {
    int size = 1073741824;

    dim3 block(64);
    dim3 grid((size + block.x - 1) / block.x, 1);

    size_t bytes = size * sizeof(int);
    int *h_inputData = (int *) malloc(bytes);
    int *tmp = (int *) malloc(bytes);
    int *h_outputData = (int *) malloc(grid.x * sizeof(int));

    for (int i = 0; i < size; i++) {
        h_inputData[i] = 1;
        tmp[i] = 1;
    }

    double iStart, iElapsed;
    int gpuSum;

    int *d_inputData;
    int *d_outputData;
    hipMalloc((void **) &d_inputData, bytes);
    hipMalloc((void **) &d_outputData, grid.x * sizeof(int));

    iStart = seconds();
    int cpu_sum = recursiveReduce(tmp, size);
    iElapsed = seconds() - iStart;
    printf("cpu reduce   elapsed time: %f ms   cpu sum: %d\n", iElapsed * 1000, cpu_sum);

    hipMemcpy(d_inputData, h_inputData, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    iStart = seconds();
    reduceInterleaved<<<grid, block>>>(d_inputData, d_outputData, size);
    hipDeviceSynchronize();
    iElapsed = seconds() - iStart;
    hipMemcpy(h_outputData, d_outputData, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpuSum = 0;
    for (int i = 0; i < grid.x; i++) gpuSum += h_outputData[i];
    printf("gpu reduce   elapsed time: %f ms   gpu sum: %d\n", iElapsed * 1000, gpuSum);

    free(h_inputData);
    free(h_outputData);
    free(tmp);
    hipFree(d_inputData);
    hipFree(d_outputData);
    hipDeviceReset();

    return 0;


}