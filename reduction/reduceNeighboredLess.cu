#include "hip/hip_runtime.h"
//
// Created by mohamadbastin on 7/27/22.
//
#include "cstdio"
#include "../common/common.cpp"


__global__ void reduceNeighboredLess(int *g_inputData, int *g_outputData, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + tid;

    if (idx > n) return;

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = tid * stride * 2;
        if (index < blockDim.x) {
            g_inputData[idx] += g_inputData[idx + stride];
        }
        __syncthreads();
    }
    if (tid == 0) {
        g_outputData[blockIdx.x] = g_inputData[idx];
    }
}

int main() {
    int size = 1073741824;

    dim3 block(64);
    dim3 grid((size + block.x - 1) / block.x, 1);

    size_t bytes = size * sizeof(int);
    int *h_inputData = (int *) malloc(bytes);
    int *tmp = (int *) malloc(bytes);
    int *h_outputData = (int *) malloc(grid.x * sizeof(int));

    for (int i = 0; i < size; i++) {
        h_inputData[i] = 1;
        tmp[i] = 1;
    }

    double iStart, iElapsed;
    int gpuSum;

    int *d_inputData;
    int *d_outputData;
    hipMalloc((void **) &d_inputData, bytes);
    hipMalloc((void **) &d_outputData, grid.x * sizeof(int));

    iStart = seconds();
    int cpu_sum = recursiveReduce(tmp, size);
    iElapsed = seconds() - iStart;
    printf("cpu reduce   elapsed time: %f ms   cpu sum: %d\n", iElapsed * 1000, cpu_sum);

    hipMemcpy(d_inputData, h_inputData, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    iStart = seconds();
    reduceNeighboredLess<<<grid, block>>>(d_inputData, d_outputData, size);
    hipDeviceSynchronize();
    iElapsed = seconds() - iStart;
    hipMemcpy(h_outputData, d_outputData, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpuSum = 0;
    for (int i = 0; i < grid.x; i++) gpuSum += h_outputData[i];
    printf("gpu reduce   elapsed time: %f ms   gpu sum: %d\n", iElapsed * 1000, gpuSum);

    free(h_inputData);
    free(h_outputData);
    free(tmp);
    hipFree(d_inputData);
    hipFree(d_outputData);
    hipDeviceReset();

    return 0;


}