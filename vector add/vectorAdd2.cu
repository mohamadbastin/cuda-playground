//
// Created by mohamadbastin on 6/29/22.
//


#include <hip/hip_runtime.h>
#include "cstdio"

void naiveVectorAdd(int *a, int *b, int *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

void initializeData(int *a, int n) {
    for (int i = 0; i < n; i++) {
        a[i] = i;
    }
}

__global__ void vectorAdd(int *a, int *b, int *c) {
    int x = threadIdx.x;
    c[x] = a[x] + b[x];

}

int main() {
    int n = 10;
    int *h_a, *h_b, *h_c;
    h_a = (int *) malloc(n * sizeof(int));
    h_b = (int *) malloc(n * sizeof(int));
    h_c = (int *) malloc(n * sizeof(int));

    initializeData(h_a, n);
    initializeData(h_b, n);

    int *d_a, *d_b, *d_c;
    hipMalloc((int **) &d_a, n * sizeof(int));
    hipMalloc((int **) &d_b, n * sizeof(int));
    hipMalloc((int **) &d_c, n * sizeof(int));

    hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);


//    naiveVectorAdd(h_a, h_b, h_c, n);
    vectorAdd<<<1, 10>>>(d_a, d_b, d_c);

    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d ", h_c[i]);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
