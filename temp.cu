//
// Created by mohamadbastin on 6/30/22.
//

#include <hip/hip_runtime.h>
#include "cstdio"

int main(int argc, char **argv) {
    int n = 1024;

    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    block.x = 512;
    grid.x = (n + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    block.x = 256;
    grid.x = (n + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    block.x = 128;
    grid.x = (n + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);


    return 0;
}