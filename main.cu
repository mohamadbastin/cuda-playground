
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void helloFromGPU() {
    int x = threadIdx.x;
    printf("hello from gpu thread %d.\n", x);
}

int main() {
    printf("hello world from cpu.\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
//    cudaDeviceSynchronize();

    return 0;
}

